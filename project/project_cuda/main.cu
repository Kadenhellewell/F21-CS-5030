/**
 * compile: nvcc main.cu -o cuda_streams
 */



#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>

using namespace std;




//These are separate structs to aid is differentiating points and vectors
struct Vector{
    float x_val;
    float y_val;
};

struct Point{
    float x_coord;
    float y_coord;
};

__device__
Vector const_vect_mult(float c, Vector v);
__device__
Vector get_v_from_field(int x_coord, int y_coord);
__device__
Vector get_v_from_field(float x_coord, float y_coord);
__device__
Vector get_v_from_field(Point p);
__device__
Vector add_vectors(Vector v1, Vector v2);
__device__
Point add_vector_point(Point p, Vector v);
__device__
Point rungeKutta(Point p, float time_step, Vector* vectors);
__device__
bool not_in_range(Point p);
__device__
Vector interpolate(Vector v1, Vector v2, int bigP, int smallP, float p);

const int data_cols = 1300;
const int data_rows = 600;
const int num_steps = 50;
const int stream_size = num_steps*data_rows*3;
const int num_vectors = data_rows * data_cols;
const int data_size = num_vectors*2;//2 floats per vector
//TODO: define tile size variables (whatever they are)

/**
 * Starting at the far left, calculate stream lines going to the right.
 * Each thread calculates one streamline. The row is determined by its id.
 * @param vectors the vector field
 * @param streams the output array - floats, a multiple of 3, in the order line_id, coordinate_x, coordinate_y
 */
__global__
void calculate_stream_lines(Vector* vectors, float* streams_d)//streams is the output
{
    //Each thread calculates one stream
    int thread_id = blockIdx.x*blockDim.x + threadIdx.x;//TODO: make sure this is correct for 1D (this is the threadId)

    float time_step = .2;
    int lineId = thread_id;
    if(lineId >= data_rows) return; //passed the bottom row
    //initialize the starting point at the beginning of each new line
    Point current{};
    current.x_coord = 0;//Each streamline starts at the far left
    current.y_coord = lineId;
    int startPoint = 0;
    if(lineId > 0) //0 is an edge case in this calculation
        startPoint = lineId*num_steps*3 + 1;
    for(int step = 0; step < num_steps*3; step++)//each 'step' fills out 3 elements of the array
    {
        if(not_in_range(current)) break;//The streamline has left the known vector field. This thread is done
        streams_d[startPoint + step] = lineId;
        streams_d[startPoint + ++step] = current.x_coord;
        streams_d[startPoint + ++step] = current.y_coord;
        current = rungeKutta(current, time_step, vectors);
    }
}


int main() {
    Vector* vectors;
    std::ifstream inFile("cyl2d_1300x600_float32[2].raw", std::ios::binary);

    //Read in from file
    float f;
    int k = 0;
    float * data = new float[data_size];
    while (inFile.read(reinterpret_cast<char*>(&f), sizeof(float)))
    {
        data[k] = f;
        k++;
    }
    //set vector objects
    for(int i = 0; i < data_size; i++)
    {
        int index = i/2;//i will always be even at this point
        Vector thisVector{};
        thisVector.x_val = data[i];
        thisVector.y_val = data[++i];
        vectors[index] = thisVector;
    }

    //Allocate spaced on the GPU for vectors, then copy up
    Vector* vectors_d;
    hipMalloc(&vectors_d, num_vectors);
    hipMemcpy(vectors_d, vectors, num_vectors, hipMemcpyHostToDevice);

    //initialize streams
    float* streams = new float[stream_size];
    for(int i = 0; i < stream_size; i++)
    {
        streams[i] = -1;
    }

    //allocate space for device results (GPU)
    float* streams_d;
    hipMalloc(&streams_d, stream_size);
    hipMemcpy(streams_d, streams, stream_size, hipMemcpyHostToDevice);

    //I think I want 1D blocks, since each streamline starts in the first column. Not sure how to do that.
    //Set grid and block sizes
    dim3 DimGrid(1, 1, 1);// how many blocks
    dim3 DimBlock(600, 1, 1);// how many threads per block

    calculate_stream_lines<<<DimGrid, DimBlock>>>(vectors_d, streams_d);
    //copy results of calculating streams to host
    hipMemcpy(streams, streams_d, stream_size, hipMemcpyDeviceToHost);

    std::ofstream outFile("streamlines_cuda.csv", std::ios::app);
    //Parse results, groups of 3, line_id, coordinate_x, coordinate_y
    outFile << "line_id, coordinate_x, coordinate_y" << endl;
    //print local streams to file
    for(int j = 0; j < stream_size; j++)
    {
        if(streams[j] != -1)
            outFile << streams[j] << ", " << streams[++j] << ", " << streams[++j] << endl;
    }
    hipFree(vectors);
    hipFree(streams_d);
    delete[] streams;
    return 0;
}

/**
 * Get the value of the given vector field at the given point.
 * For integers, this can be directly retrieved from the vector field (no interpolation needed).
 * @param x_coord the x coordinate
 * @param y_coord the y coordinate
 * @param vectors the vector field
 * @return a Vector object
 */
__device__
Vector get_v_from_field(int x_coord, int y_coord, Vector* vectors)
{
    int index = y_coord*data_cols + x_coord;
    return vectors[index];
}

/**
 * Get the value of the given vector field at the given point.
 * This method uses Bilinear Interpolation.
 * @param x_coord the x coordinate
 * @param y_coord the y coordinate
 * @param vectors the vector field
 * @return a Vector object
 */
__device__
Vector get_v_from_field(float x_coord, float y_coord, Vector* vectors)
{
    //Bilinear Interpolation
    //Get integer points around the given x and y
    int floor_y = floor(y_coord);
    int floor_x = floor(x_coord);
    int ceil_y = ceil(y_coord);
    int ceil_x = ceil(x_coord);

    if(ceil_x == floor_x && ceil_y == floor_y)//both are integers, no interpolation
    {
        return get_v_from_field((int)x_coord, (int)y_coord, vectors);
    }

    Vector R1{}, R2{};
    //Linear interpolation
    if(ceil_x == floor_x) //x is an integer, y is not
    {
        R1 = get_v_from_field((int)x_coord, floor_y);
        R2 = get_v_from_field((int)x_coord, ceil_y);
        return interpolate(R1, R2, ceil_y, floor_y, y_coord);
    }

    if(ceil_y == floor_y)//y is an integer, x is not
    {
        R1 = get_v_from_field(floor_x, (int)y_coord);
        R2 = get_v_from_field(ceil_x, (int)y_coord);
        return interpolate(R1, R2, ceil_x, floor_x, x_coord);
    }

    //Neither are integers
    //bilinear interpolation
    //Q11 - bottom left; Q12 - top left; Q21 - bottom right; Q22 - top right
    Vector Q11 = get_v_from_field(floor_x, floor_y, vectors);
    Vector Q12 = get_v_from_field(floor_x, ceil_y, vectors);
    Vector Q21 = get_v_from_field(ceil_x, floor_y, vectors);
    Vector Q22 = get_v_from_field(ceil_x, ceil_y, vectors);

    //Calculate R10
    R1 = interpolate(Q11, Q21, ceil_x, floor_x, x_coord);

    //Calculate R2
    R2 = interpolate(Q12, Q22, ceil_x, floor_x, x_coord);

    //Calculate P
    return interpolate(R1, R2, ceil_y, floor_y, y_coord);
}
/*
 * How to calculate R1, R2, and P (from https://x-engineer.org/bilinear-interpolation/)
 * \[R_{1}(x, y) = Q_{11} \frac{x_{2}-x}{x_{2}-x_{1}} + Q_{21} \frac{x-x_{1}}{x_{2}-x_{1}} \tag{1}\]

   \[R_{2}(x, y) = Q_{12} \frac{x_{2}-x}{x_{2}-x_{1}} + Q_{22} \frac{x-x_{1}}{x_{2}-x_{1}} \tag{2}\]

   \[{P(x,y) = R_{1} \frac{y_{2}-y}{y_{2}-y_{1}} + R_{2} \frac{y-y_{1}}{y_{2}-y_{1}}} \tag{3}\]
 */

/**
 * This method performs linear interpolation.
 * @param v1 the vector corresponding to the larger point
 * @param v2 the vector corresponding to the smaller point
 * @param bigP the larger point
 * @param smallP the smaller point
 * @param p the desired point
 * @return the vector at the desired point
 */
__device__
Vector interpolate(Vector v1, Vector v2, int bigP, int smallP, float p)
{
    Vector temp1 = const_vect_mult((bigP - p) / (bigP - smallP), v1);
    Vector temp2 = const_vect_mult((p - smallP) / (bigP - smallP), v2);
    Vector returnVector = add_vectors(temp1, temp2);
    return returnVector;
}

/**
 * Wrapper function. Gets the vector associated with a point.
 * @param p the point
 * @param vectors the vector field
 * @return the desired vector
 */
__device__
Vector get_v_from_field(Point p, Vector* vectors)
{
    return get_v_from_field(p.x_coord, p.y_coord, vectors);
}

/**
 * Multiply a vector by a constant.
 * @param c the constant
 * @param v the vector
 * @return the new vector
 */
__device__
Vector const_vect_mult(float c, Vector v)
{
    Vector returnVector{};
    returnVector.x_val = c*v.x_val;
    returnVector.y_val = c*v.y_val;
    return returnVector;
}

/**
 * Add 2 vectors together
 * @param v1 the first vector
 * @param v2 the second vector
 * @return the sum of the two vectors
 */
__device__
Vector add_vectors(Vector v1, Vector v2)
{
    Vector returnVector{};
    returnVector.x_val = v1.x_val + v2.x_val;
    returnVector.y_val = v1.y_val + v2.y_val;
    return returnVector;
}

/**
 * Add a vector to a point to get a new point
 * @param p the starting point
 * @param v the vector
 * @return the new point
 */
__device__
Point add_vector_point(Point p, Vector v)
{
    Point returnPoint{};
    returnPoint.x_coord = p.x_coord + v.x_val;
    returnPoint.y_coord = p.y_coord + v.y_val;
    return returnPoint;
}

/**
 * Do the Runge-Kutta algorithm
 * @param p the starting point
 * @param time_step the time step
 * @param vectors the vector field
 * @return the next point
 */
__device__
Point rungeKutta(Point p, float time_step, Vector* vectors)
{
    Vector k1{}, k2{}, k3{}, k4{};
    Point failPoint{};
    failPoint.x_coord = -1;
    failPoint.y_coord = -1;

    // Apply Runge Kutta Formulas
    // to find next value of y
    k1 = const_vect_mult(time_step, get_v_from_field(p));
    Point p1 = add_vector_point(p, const_vect_mult(.5, k1));
    if(not_in_range(p1)) return failPoint;
    Vector v_1 = get_v_from_field(p1, vectors);

    k2 = const_vect_mult(time_step, v_1);
    Point p2 = add_vector_point(p, const_vect_mult(.5, k2));
    if(not_in_range(p2)) return failPoint;
    Vector v_2 = get_v_from_field(p2, vectors);

    k3 = const_vect_mult(time_step, v_2);
    Point p3 = add_vector_point(p, k3);
    if(not_in_range(p3)) return failPoint;
    Vector v_3 = get_v_from_field(p3, vectors);

    k4 = const_vect_mult(time_step, v_3);
    Vector tempSum = k1;
    tempSum = add_vectors(tempSum, const_vect_mult(2, k2));
    tempSum = add_vectors(tempSum, const_vect_mult(2, k3));
    tempSum = add_vectors(tempSum, k4);

    Vector temp = const_vect_mult(0.1667, tempSum);
    Point nextPoint = add_vector_point(p, temp);
    return nextPoint;
}
//Algorithm from: https://web.cs.ucdavis.edu/~ma/ECS177/papers/particle_tracing.pdf

/**
 * Check if a point is within the given vector field
 * @param p the point
 * @return whether the point is not in the vector field
 */
__device__
bool not_in_range(Point p)
{
    return p.x_coord < 0 || p.x_coord >= data_cols || p.y_coord < 0 || p.y_coord >= data_rows;
}

/**
 * Get input from the user, store, and broadcast
 * @param argc number of arguments
 * @param argv array containing the arguments
 */
void get_args(int argc, char* argv[])
{

}




