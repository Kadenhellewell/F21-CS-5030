#include "hip/hip_runtime.h"
/**
 * compile: nvcc main.cu -o cuda_streams
 */


#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>

using namespace std;




//These are separate structs to aid is differentiating points and vectors
struct Vector{
    float x_val;
    float y_val;
};

struct Point{
    float x_coord;
    float y_coord;
};



void get_args(int argc, char* argv[]);

Vector const_vect_mult(float c, Vector v);
Vector get_v_from_field(int x_coord, int y_coord);
Vector get_v_from_field(float x_coord, float y_coord);
Vector get_v_from_field(Point p);
Vector add_vectors(Vector v1, Vector v2);
Point add_vector_point(Point p, Vector v);
Point rungeKutta(Point p, float time_step);
bool not_in_range(Point p);
Vector interpolate(Vector v1, Vector v2, int bigP, int smallP, float p);

//I don't know if I can do this variables...
__global__
int data_cols = 1300;
__global__
int data_rows = 600;
//for a total of 780,000 vectors
__global__
int num_steps = 50;
//TODO: define tile size variables (whatever they are)

__device__
void calculate_stream_lines(Vector* vectors, float* streams)//streams is the output
{
    //Each thread calculates one stream
    int i = blockIdx.x*blockDim.x + threadIdx.x;//TODO: make sure this is correct for 1D
    int startPoint = i*num_steps*3;//TODO: calculate this
    Point current{};
    current.x_coord = initial_x;
    current.y_coord = initial_y;
    if(lineId >= data_cols) break; //passed the bottom row
    current.x_coord = 0;
    current.y_coord = lineId;
    for(int step = 0; step < num_steps; step++)
    {
        if(not_in_range(current)) break;//The streamline has left the known vector field. Go to the next line.i
        streams[startPoint] = lineId;
        streams[startPoint + (++step)] = current.x_coord;
        streams[startPoint + (+=step)] = current.y_coord;
        current = rungeKutta(current, time_step);
    }
}


int main() {
    Vector* vectors;
    std::ifstream inFile("cyl2d_1300x600_float32[2].raw", std::ios::binary);
    std::ofstream outFile("streamlines_cuda.csv", std::ios::app);
    int stream_size = num_steps*data_rows*3;//data_rows streams, num_steps per stream, 3 floats per step: line_id, coordinate_x, coordinate_y

    float * buffer = nullptr;
    int size;
    if (inFile) {
        // get length of file:
        inFile.seekg (0, inFile.end);
        int length = inFile.tellg();
        inFile.seekg (0, inFile.beg);
        size = length / sizeof(float);
        buffer = new float[size];

        std::cout << "Reading " << length << " characters... " << endl;
        // read data as a block:
        inFile.read ((char*)buffer, length);

        if (inFile)
            std::cout << "all characters read successfully." << endl;
        else
            std::cout << "error: only " << inFile.gcount() << " could be read";
        inFile.close();


        //Set up vector of vectors
        for(unsigned int i = 0; i < size; i++)
        {
            Vector thisVector{};
            thisVector.x_val = buffer[i];
            thisVector.y_val = buffer[++i];
            vectors[i] = thisVector;
        }
        //Allocate spaced on the GPU for vectors, then copy up
        hipMalloc(vectors, size);
        Vector* vectors_d;
        hipMemcpy(vectors_d, vectors, size, hipMemcpyHostToDevice);
    }
    //allocate space for device results (GPU)
    float* streams_d;
    hipMalloc(streams_d, stream_size);

    //I think I want 1D blocks, since each streamline starts in the first column. Not sure how to do that.
    //Set grid and block sizes
    dim3 dimGrid;
    dim3 dimBlock;

    calculate_stream_lines<<<dimGrid, dimBlock>>>(vectors, streams_d);
    //copy results of calculating streams to host
    float* results;
    hipMemcpy(results, streams_d, stream_size, hipMemcpyDeviceToHost);
    //Parse results, groups of 3, line_id, coordinate_x, coordinate_y
    outFile << "line_id, coordinate_x, coordinate_y" << endl;
    for(int line = 0; line < stream_size; line++)
    {
        outFile << results[line] << ", " << results[++line] << ", " << results[++line] << endl;
    }
    return 0;
}

/**
 * Get the value of the given vector field at the given point.
 * For integers, this can be directly retrieved from the vector field (no interpolation needed).
 * @param x_coord the x coordinate
 * @param y_coord the y coordinate
 * @param vectors the vector field
 * @return a Vector object
 */
__device__
Vector get_v_from_field(int x_coord, int y_coord, Vector* vectors)
{
    int index = y_coord*data_cols + x_coord;
    return vectors[index];
}

/**
 * Get the value of the given vector field at the given point.
 * This method uses Bilinear Interpolation.
 * @param x_coord the x coordinate
 * @param y_coord the y coordinate
 * @param vectors the vector field
 * @return a Vector object
 */
__device__
Vector get_v_from_field(float x_coord, float y_coord, Vector* vectors)
{
    //Bilinear Interpolation
    //Get integer points around the given x and y
    int floor_y = floor(y_coord);
    int floor_x = floor(x_coord);
    int ceil_y = ceil(y_coord);
    int ceil_x = ceil(x_coord);

    if(ceil_x == floor_x && ceil_y == floor_y)//both are integers, no interpolation
    {
        return get_v_from_field((int)x_coord, (int)y_coord, vectors);
    }

    Vector R1{}, R2{};
    //Linear interpolation
    if(ceil_x == floor_x) //x is an integer, y is not
    {
        R1 = get_v_from_field((int)x_coord, floor_y);
        R2 = get_v_from_field((int)x_coord, ceil_y);
        return interpolate(R1, R2, ceil_y, floor_y, y_coord);
    }

    if(ceil_y == floor_y)//y is an integer, x is not
    {
        R1 = get_v_from_field(floor_x, (int)y_coord);
        R2 = get_v_from_field(ceil_x, (int)y_coord);
        return interpolate(R1, R2, ceil_x, floor_x, x_coord);
    }

    //Neither are integers
    //bilinear interpolation
    //Q11 - bottom left; Q12 - top left; Q21 - bottom right; Q22 - top right
    Vector Q11 = get_v_from_field(floor_x, floor_y, vectors);
    Vector Q12 = get_v_from_field(floor_x, ceil_y, vectors);
    Vector Q21 = get_v_from_field(ceil_x, floor_y, vectors);
    Vector Q22 = get_v_from_field(ceil_x, ceil_y), vectors;

    //Calculate R10
    R1 = interpolate(Q11, Q21, ceil_x, floor_x, x_coord);

    //Calculate R2
    R2 = interpolate(Q12, Q22, ceil_x, floor_x, x_coord);

    //Calculate P
    return interpolate(R1, R2, ceil_y, floor_y, y_coord);
}
/*
 * How to calculate R1, R2, and P (from https://x-engineer.org/bilinear-interpolation/)
 * \[R_{1}(x, y) = Q_{11} \frac{x_{2}-x}{x_{2}-x_{1}} + Q_{21} \frac{x-x_{1}}{x_{2}-x_{1}} \tag{1}\]

   \[R_{2}(x, y) = Q_{12} \frac{x_{2}-x}{x_{2}-x_{1}} + Q_{22} \frac{x-x_{1}}{x_{2}-x_{1}} \tag{2}\]

   \[{P(x,y) = R_{1} \frac{y_{2}-y}{y_{2}-y_{1}} + R_{2} \frac{y-y_{1}}{y_{2}-y_{1}}} \tag{3}\]
 */

/**
 * This method performs linear interpolation.
 * @param v1 the vector corresponding to the larger point
 * @param v2 the vector corresponding to the smaller point
 * @param bigP the larger point
 * @param smallP the smaller point
 * @param p the desired point
 * @return the vector at the desired point
 */
__device__
Vector interpolate(Vector v1, Vector v2, int bigP, int smallP, float p)
{
    Vector temp1 = const_vect_mult((bigP - p) / (bigP - smallP), v1);
    Vector temp2 = const_vect_mult((p - smallP) / (bigP - smallP), v2);
    Vector returnVector = add_vectors(temp1, temp2);
    return returnVector;
}

/**
 * Wrapper function. Gets the vector associated with a point.
 * @param p the point
 * @param vectors the vector field
 * @return the desired vector
 */
__device__
Vector get_v_from_field(Point p, Vector* vectors)
{
    return get_v_from_field(p.x_coord, p.y_coord, vectors);
}

/**
 * Multiply a vector by a constant.
 * @param c the constant
 * @param v the vector
 * @return the new vector
 */
__device__
Vector const_vect_mult(float c, Vector v)
{
    Vector returnVector{};
    returnVector.x_val = c*v.x_val;
    returnVector.y_val = c*v.y_val;
    return returnVector;
}

/**
 * Add 2 vectors together
 * @param v1 the first vector
 * @param v2 the second vector
 * @return the sum of the two vectors
 */
__device__
Vector add_vectors(Vector v1, Vector v2)
{
    Vector returnVector{};
    returnVector.x_val = v1.x_val + v2.x_val;
    returnVector.y_val = v1.y_val + v2.y_val;
    return returnVector;
}

/**
 * Add a vector to a point to get a new point
 * @param p the starting point
 * @param v the vector
 * @return the new point
 */
__device__
Point add_vector_point(Point p, Vector v)
{
    Point returnPoint{};
    returnPoint.x_coord = p.x_coord + v.x_val;
    returnPoint.y_coord = p.y_coord + v.y_val;
    return returnPoint;
}

/**
 * Do the Runge-Kutta algorithm
 * @param p the starting point
 * @param time_step the time step
 * @param vectors the vector field
 * @return the next point
 */
__device__
Point rungeKutta(Point p, float time_step, Vector* vectors)
{
    Vector k1{}, k2{}, k3{}, k4{};

    // Apply Runge Kutta Formulas
    // to find next value of y
    k1 = const_vect_mult(time_step, get_v_from_field(p));
    Point p1 = add_vector_point(p, const_vect_mult(.5, k1));
    Vector v_1 = get_v_from_field(p1, vectors);

    k2 = const_vect_mult(time_step, v_1);
    Point p2 = add_vector_point(p, const_vect_mult(.5, k2));
    Vector v_2 = get_v_from_field(p2, vectors);

    k3 = const_vect_mult(time_step, v_2);
    Point p3 = add_vector_point(p, k3);
    Vector v_3 = get_v_from_field(p3, vectors);

    k4 = const_vect_mult(time_step, v_3);
    Vector tempSum = k1;
    tempSum = add_vectors(tempSum, const_vect_mult(2, k2));
    tempSum = add_vectors(tempSum, const_vect_mult(2, k3));
    tempSum = add_vectors(tempSum, k4);

    Vector temp = const_vect_mult(0.1667, tempSum);
    Point nextPoint = add_vector_point(p, temp);
    return nextPoint;
}
//Algorithm from: https://web.cs.ucdavis.edu/~ma/ECS177/papers/particle_tracing.pdf

/**
 * Check if a point is within the given vector field
 * @param p the point
 * @return whether the point is not in the vector field
 */
__device__
bool not_in_range(Point p)
{
    return p.x_coord < 0 || p.x_coord >= data_cols || p.y_coord < 0 || p.y_coord >= data_rows;
}

/**
 * Get input from the user, store, and broadcast
 * @param argc number of arguments
 * @param argv array containing the arguments
 */
void get_args(int argc, char* argv[])
{

}




